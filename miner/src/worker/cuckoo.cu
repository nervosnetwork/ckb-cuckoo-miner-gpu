#include "hip/hip_runtime.h"
﻿// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stddef.h>
#include <openssl/rand.h>
#include "blake2b.h"
#include <time.h>

#define threadsPerBlock  (512)
#define MaxCuckooNum (4*4096)
#define trim (32)
#define SolveThreadsPerBlock (128)
#define SolveEN (128)
#define CuckooNum (4*4096)

#define rotl(x, b) (((x) << (b)) | ((x) >> (64 - (b))))
#define EBIT 15
#define CLEN 12
#define EN (1 << EBIT)
#define M (EN << 1)
#define MASK ((1 << EBIT) - 1)
#define CN CLEN << 2

uint32_t cproof[CuckooNum][CLEN] = { 0 };
uint8_t msg[CuckooNum][32] = { 0 };
uint8_t alive[CuckooNum][EN >> 3] = { 0 };
uint8_t calive[CuckooNum][EN >> 3] = { 0 };
uint64_t nonces[CuckooNum];

uint8_t  *gmsg = NULL;
uint8_t  *gRHash = NULL;
uint32_t *gRege = NULL;
uint32_t *gproof = NULL;
uint32_t *gnode = NULL;


// set siphash keys from 32 byte char array
#define setkeys() \
	k0 = (((uint64_t *)mesg)[0]); \
	k1 = (((uint64_t *)mesg)[1]); \
	k2 = (((uint64_t *)mesg)[2]); \
	k3 = (((uint64_t *)mesg)[3]);

#define sip_round() \
	v0 += v1; v2 += v3; v1 = rotl(v1, 13); \
	v3 = rotl(v3, 16); v1 ^= v0; v3 ^= v2; \
	v0 = rotl(v0, 32); v2 += v1; v0 += v3; \
	v1 = rotl(v1, 17); v3 = rotl(v3, 21); \
	v1 ^= v2; v3 ^= v0; v2 = rotl(v2, 32);

#define siphash24( nonce , hashv ) { \
	v0 = k0; v1 = k1; v2 = k2; v3 = k3; \
	v3 ^= (nonce); \
	sip_round(); sip_round(); \
	v0 ^= (nonce); \
	v2 ^= 0xff; \
	sip_round(); sip_round(); sip_round(); sip_round(); \
	hashv = (v0 ^ v1 ^ v2  ^ v3); \
	}

__global__ void kill_leaf(uint8_t *gmesg, uint32_t *gRege, uint8_t *gRHash, uint32_t *gnode)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	uint64_t k0, k1, k2, k3;
	uint64_t v0, v1, v2, v3;
	uint64_t hash;
	uint64_t st, ed;
	uint32_t tmp, tw_bit, py,py2;
	uint32_t i,j;
	uint32_t u;
	uint32_t uorv,edgeidx;

	uint32_t block_tid = id % threadsPerBlock;
	uint32_t block_id = id / threadsPerBlock;
	uint32_t block_ENrange = EN / threadsPerBlock;
	uint32_t block_AliveRange = (EN >> 3) / threadsPerBlock;
	uint32_t block_AliveNode = (EN >> 4) / threadsPerBlock;
	uint8_t *mesg = gmesg + block_id * 32;

	uint32_t *node32 = gnode + block_id * M;

	__shared__ uint8_t alive_ege[EN >> 3];
	__shared__ uint32_t node[EN >> 4];
	__shared__ uint32_t RHash[SolveEN][2];
	__shared__ uint32_t RegeSP;

	if (block_tid == 0)atomicAnd(&RegeSP,0);

	setkeys();
	st = block_tid*block_ENrange; ed = (block_tid + 1)*block_ENrange;

	memset(alive_ege + block_tid*block_AliveRange, 0, block_AliveRange);
	__syncthreads();
	for (i = st; i < ed; i++)
	{
		siphash24((i << 1) + 0, hash);
		u = (hash & MASK);
		node32[i << 1] = u;
		siphash24((i << 1) + 1, hash);
		u = (hash & MASK);
		node32[(i << 1) +1] = u;
	}
	__syncthreads();

	for (j = 0; j < trim; j++)
	{
		uorv = 0;
		memset(node + block_tid*block_AliveNode, 0, block_AliveNode*sizeof(uint32_t));__syncthreads();

		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				py = 1 << ((u << 1) & 31);
				tmp = atomicOr(&node[u >> 4], py);
				py2 = py << 1;
				if ((tmp & (py2 | py)) == py)atomicOr(&node[u >> 4], py2);
			}
		}
		__syncthreads();
		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{		
				u = node32[(i << 1) + uorv];
				tmp = node[u >> 4];
				py = ((u << 1) & 31);
				tw_bit = (tmp >> py) & 2;
				if (!tw_bit)
				{
					alive_ege[i >> 3] = alive_ege[i >> 3] ^ (1 << (i & 7));
				}
			}
		}
		__syncthreads();
		uorv = 1;
		memset(node + block_tid*block_AliveNode, 0, block_AliveNode*sizeof(uint32_t)); __syncthreads();

		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				py = 1 << ((u << 1) & 31);
				tmp = atomicOr(&node[u >> 4], py);
				py2 = py << 1;
				if ((tmp & (py2 | py)) == py)atomicOr(&node[u >> 4], py2);
			}
		}
		__syncthreads();
		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				tmp = node[u >> 4];
				py = ((u << 1) & 31);
				tw_bit = (tmp >> py) & 2;
				if (!tw_bit)
				{
					alive_ege[i >> 3] = alive_ege[i >> 3] ^ (1 << (i & 7));
				}
			}
		}
		__syncthreads();
	}
	__syncthreads();

	gRege[SolveEN*block_id + block_tid % SolveEN] = 0xffffffff;
	__syncthreads();
	for (i = st; i < ed; i++)
	{
		if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
		{
			edgeidx = atomicInc(&RegeSP, 126);
			gRege[SolveEN * block_id + edgeidx] = i;
			u = node32[(i << 1) + 0]<<1;
			RHash[edgeidx][0] = u;
			u = node32[(i << 1) + 1] << 1 + 1;
			RHash[edgeidx][1] = u;
		}
	}
	__syncthreads();

	if (block_tid<=1)
	{
		tmp = 0;
		edgeidx = RegeSP;
		for (i = 0; i < edgeidx;i++)
		{
			py = RHash[i][block_tid];
			if (py == 0xffffffff)continue;
			RHash[i][block_tid] = 0xffffffff;
			gRHash[(SolveEN << 1) *block_id + (i << 1) + block_tid] = (tmp << 1) + block_tid;
			for (j = i+1; j < edgeidx; j++)
			{
				py2 = RHash[j][block_tid];
				if (py2 == py)
				{
					RHash[j][block_tid] = 0xffffffff;
					gRHash[(SolveEN << 1) *block_id + (j << 1) + block_tid] = (tmp << 1) + block_tid;
				}
			}
			tmp++;
		}
	}
	__syncthreads();

}

__global__ void solve128X_127EN(uint32_t *gRege, uint8_t *gRHash, uint32_t *gproof)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t i,tmp;
	uint8_t u, v;

	uint32_t block_tid = id % SolveThreadsPerBlock;
	uint32_t *Rege = gRege + id * SolveEN;
	uint8_t *RHash = gRHash + id * (SolveEN << 1);
	uint32_t *proof = gproof + id * CLEN;

	__shared__ uint32_t path[SolveThreadsPerBlock][CLEN];
	__shared__ uint8_t graph[SolveThreadsPerBlock][SolveEN<<1];

	uint8_t pre;
	uint8_t cur;
	uint8_t next;

	memset(&graph[block_tid], 0xff, (SolveEN << 1));
	proof[0] = 0xffffffff;

	for (i = 0; i<SolveEN; i++)
	{
		if (Rege[i] == 0xffffffff)
		{
			break;
		}
		u = RHash[i<<1];
		v = RHash[(i << 1)+1];
		__syncthreads();
		pre = 0xff;
		cur = u;
		while (cur != 0xff)
		{
			next = graph[block_tid][cur];
			graph[block_tid][cur] = pre;
			pre = cur;
			cur = next;
		}
		int m = 0;
		cur = v;
		while (graph[block_tid][cur] != 0xff && m < CLEN)
		{
			cur = graph[block_tid][cur];
			++m;
		}
		if (cur != u)
		{
			graph[block_tid][u] = v;
		}
		else if (m == CLEN - 1)
		{
			int j;
			cur = v;
			for (j = 0; j <= m; ++j)
			{
				path[block_tid][j] = cur;
				cur = graph[block_tid][cur];
			}

			memset(&graph[block_tid], 0xff, (SolveEN << 1));

			for (j = 1; j <= m; ++j)
			{
				graph[block_tid][path[block_tid][j]] = path[block_tid][j - 1];
			}

			int k = 0;
			int b = CLEN - 1;
			for (j = 0; k < b; ++j)
			{
				u = RHash[j << 1];
				v = RHash[(j << 1) + 1];
				if (graph[block_tid][u] == v || graph[block_tid][v] == u)
				{
					path[block_tid][k] = Rege[j];
					++k;
				}
			}
			path[block_tid][k] = Rege[i];

			for (j = 0; j < CLEN-1; j++) // sort
			{
				for (k = 0; k < CLEN-j-1; k++)
				{
					if (path[block_tid][k]>path[block_tid][k+1])
					{
						tmp = path[block_tid][k];
						path[block_tid][k] = path[block_tid][k + 1];
						path[block_tid][k + 1] = tmp;
					}
				}
			}
			for (j = 0; j < CLEN; j++)proof[j] = path[block_tid][j];
			break;
		}
	}
	__syncthreads();
}

int gpu_cuckoo()
{
	if (CuckooNum > MaxCuckooNum) { 
		printf("CuckooNum out of bound!!!\n");
		return 0; 
	}

	if (CuckooNum % SolveThreadsPerBlock != 0) {
		printf("CuckooNum must be a multiple of SolveThreadsPerBlock = %5d\n", SolveThreadsPerBlock);
		return 0;
	}
	
	//alloc once
	if (gmsg == NULL) {
		if (hipMalloc((void **)&gmsg, CuckooNum * 32 * sizeof(uint8_t)) != hipSuccess) {
			printf("gpwd hipMalloc error\n");
			return 0;
		}
	}

	if (gRege == NULL) {
		if (hipMalloc((void **)&gRege, CuckooNum * 128 * sizeof(uint32_t)) != hipSuccess) {
			printf("gpwd hipMalloc error\n");
			return 0;
		}
	}

	if (gRHash == NULL) {
		if (hipMalloc((void **)&gRHash, CuckooNum * 2 * 128 * sizeof(uint8_t)) != hipSuccess) {
			printf("gpwd hipMalloc error\n");
			return 0;
		}
	}

	if (gproof == NULL) {
		if (hipMalloc((void **)&gproof, CuckooNum * CLEN * sizeof(uint32_t)) != hipSuccess) {
			printf("gpwd hipMalloc error\n");
			return 0;
		}
	}

	if (gnode == NULL) {
		if (hipMalloc((void **)&gnode, CuckooNum * M * sizeof(uint32_t)) != hipSuccess) {
			printf("gpwd hipMalloc error\n");
			return 0;
		}
	}

	if (hipMemcpy(gmsg, msg, CuckooNum * 32 * sizeof(uint8_t), hipMemcpyHostToDevice) != hipSuccess) {
		printf("copy memory error\n");
		return 0;
	}
	
	kill_leaf << <CuckooNum, threadsPerBlock >> >(gmsg, gRege, gRHash, gnode);
	hipDeviceSynchronize();

	solve128X_127EN << <CuckooNum / SolveThreadsPerBlock, SolveThreadsPerBlock >> >(gRege, gRHash, gproof);
	hipDeviceSynchronize();
	
	if (hipMemcpy(cproof, gproof, CuckooNum * CLEN * sizeof(uint32_t), hipMemcpyDeviceToHost) != hipSuccess) {
		printf("copy memory error\n");
		return 0;
	}
	

	return CuckooNum;
}

extern "C" {
	int c_solve(uint32_t *prof, uint64_t *nonc, const uint8_t *hash, const uint8_t *target) {
		uint8_t pmesg[CN];
		uint8_t thash[32];
		blake2b_state S;
		
		b2setup(&S);
		memcpy(pmesg+8, hash, 32);

		for(int i=0; i< CuckooNum; ++i) {
			RAND_bytes(pmesg, 8);
			blake2b_state tmp = S;
			blake2b_update(&tmp, pmesg, 40);
			blake2b_final(&tmp, msg[i], 32);
			nonces[i] = le64toh(((uint64_t *)pmesg)[0]);
		}

		int ret = gpu_cuckoo();

		for(int i=0; i< ret; ++i) {
			if (cproof[i][0] != 0xffffffff)
			{
				memcpy(pmesg, cproof[i], CN);
				blake2b_state tmp = S;
				blake2b_update(&tmp, pmesg, CN);
				blake2b_final(&tmp, thash, 32);

				for(int j=0; j<32; ++j) {
					if(thash[j] < target[j]) {
						memcpy(prof, cproof[i], CN);
						*nonc = nonces[i];
						prof[CLEN] = 1;
						return ret;
					} else if(thash[j] > target[j]) {
						break;
					}
				}
			}
		}

		return ret;
	}
}
