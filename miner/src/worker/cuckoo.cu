#include "hip/hip_runtime.h"
﻿/// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stddef.h>
#include <openssl/rand.h>
#include "blake2b.h"

#define threadsPerBlock  (512)
#define MaxCuckooNum (4*4096)
#define MaxGpuNum (1024)
#define trim (32)
#define SolveThreadsPerBlock (128)
#define SolveEN (128)
#define CuckooNum (4*4096)

#define rotl(x, b) (((x) << (b)) | ((x) >> (64 - (b))))
#define EBIT 15
#define CLEN 12
#define EN (1 << EBIT)
#define M (EN << 1)
#define MASK ((1 << EBIT) - 1)
#define CN CLEN << 2

struct GPU_DEVICE
{
	uint32_t cproof[CuckooNum][CLEN];
	uint8_t msg[CuckooNum][32];
	uint8_t alive[CuckooNum][EN >> 3];
	uint8_t calive[CuckooNum][EN >> 3];
	uint64_t nonces[CuckooNum];

	uint8_t  *gmsg = NULL;
	uint8_t  *gRHash = NULL;
	uint32_t *gRege = NULL;
	uint32_t *gproof = NULL;
	uint32_t *gnode = NULL;
};

GPU_DEVICE *gpu_divices[MaxGpuNum] = {NULL};
uint32_t gpu_divices_cnt = 0;

// set siphash keys from 32 byte char array
#define setkeys() \
	k0 = (((uint64_t *)mesg)[0]); \
	k1 = (((uint64_t *)mesg)[1]); \
	k2 = (((uint64_t *)mesg)[2]); \
	k3 = (((uint64_t *)mesg)[3]);

#define sip_round() \
	v0 += v1; v2 += v3; v1 = rotl(v1, 13); \
	v3 = rotl(v3, 16); v1 ^= v0; v3 ^= v2; \
	v0 = rotl(v0, 32); v2 += v1; v0 += v3; \
	v1 = rotl(v1, 17); v3 = rotl(v3, 21); \
	v1 ^= v2; v3 ^= v0; v2 = rotl(v2, 32);

#define siphash24( nonce , hashv ) { \
	v0 = k0; v1 = k1; v2 = k2; v3 = k3; \
	v3 ^= (nonce); \
	sip_round(); sip_round(); \
	v0 ^= (nonce); \
	v2 ^= 0xff; \
	sip_round(); sip_round(); sip_round(); sip_round(); \
	hashv = (v0 ^ v1 ^ v2  ^ v3); \
}

__global__ void kill_leaf(uint8_t *gmesg, uint32_t *gRege, uint8_t *gRHash, uint32_t *gnode)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	uint64_t k0, k1, k2, k3;
	uint64_t v0, v1, v2, v3;
	uint64_t hash;
	uint64_t st, ed;
	uint32_t tmp, tw_bit, py, py2;
	uint32_t i, j;
	uint32_t u;
	uint32_t uorv, edgeidx;

	uint32_t block_tid = id % threadsPerBlock;
	uint32_t block_id = id / threadsPerBlock;
	uint32_t block_ENrange = EN / threadsPerBlock;
	uint32_t block_AliveRange = (EN >> 3) / threadsPerBlock;
	uint32_t block_AliveNode = (EN >> 4) / threadsPerBlock;
	uint8_t *mesg = gmesg + block_id * 32;

	uint32_t *node32 = gnode + block_id * M;

	__shared__ uint8_t alive_ege[EN >> 3];
	__shared__ uint32_t node[EN >> 4];
	__shared__ uint32_t RHash[SolveEN][2];
	__shared__ uint32_t RegeSP;

	if (block_tid == 0)atomicAnd(&RegeSP, 0);

	setkeys();
	st = block_tid*block_ENrange; ed = (block_tid + 1)*block_ENrange;

	memset(alive_ege + block_tid*block_AliveRange, 0, block_AliveRange);
	__syncthreads();
	for (i = st; i < ed; i++)
	{
		siphash24((i << 1) + 0, hash);
		u = (hash & MASK);
		node32[i << 1] = u;
		siphash24((i << 1) + 1, hash);
		u = (hash & MASK);
		node32[(i << 1) + 1] = u;
	}
	__syncthreads();

	for (j = 0; j < trim; j++)
	{
		uorv = 0;
		memset(node + block_tid*block_AliveNode, 0, block_AliveNode*sizeof(uint32_t)); __syncthreads();

		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				py = 1 << ((u << 1) & 31);
				tmp = atomicOr(&node[u >> 4], py);
				py2 = py << 1;
				if ((tmp & (py2 | py)) == py)atomicOr(&node[u >> 4], py2);
			}
		}
		__syncthreads();
		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				tmp = node[u >> 4];
				py = ((u << 1) & 31);
				tw_bit = (tmp >> py) & 2;
				if (!tw_bit)
				{
					alive_ege[i >> 3] = alive_ege[i >> 3] ^ (1 << (i & 7));
				}
			}
		}
		__syncthreads();
		uorv = 1;
		memset(node + block_tid*block_AliveNode, 0, block_AliveNode*sizeof(uint32_t)); __syncthreads();

		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				py = 1 << ((u << 1) & 31);
				tmp = atomicOr(&node[u >> 4], py);
				py2 = py << 1;
				if ((tmp & (py2 | py)) == py)atomicOr(&node[u >> 4], py2);
			}
		}
		__syncthreads();
		for (i = st; i < ed; i++)
		{
			if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
			{
				u = node32[(i << 1) + uorv];
				tmp = node[u >> 4];
				py = ((u << 1) & 31);
				tw_bit = (tmp >> py) & 2;
				if (!tw_bit)
				{
					alive_ege[i >> 3] = alive_ege[i >> 3] ^ (1 << (i & 7));
				}
			}
		}
		__syncthreads();
	}
	__syncthreads();

	gRege[SolveEN*block_id + block_tid % SolveEN] = 0xffffffff;
	__syncthreads();
	for (i = st; i < ed; i++)
	{
		if (!((alive_ege[i >> 3] >> (i & 7)) & 1))
		{
			edgeidx = atomicInc(&RegeSP, 0xffffffff)%126;
			gRege[SolveEN * block_id + edgeidx] = i;
			u = node32[(i << 1) + 0] << 1;
			RHash[edgeidx][0] = u;
			u = node32[(i << 1) + 1] << 1 + 1;
			RHash[edgeidx][1] = u;
		}
	}
	__syncthreads();

	if (block_tid <= 1)
	{
		tmp = 0;
		if (RegeSP > 126)edgeidx = 126;
		else	edgeidx = RegeSP;
		for (i = 0; i < edgeidx; i++)
		{
			py = RHash[i][block_tid];
			if (py == 0xffffffff)continue;
			RHash[i][block_tid] = 0xffffffff;
			gRHash[(SolveEN << 1) *block_id + (i << 1) + block_tid] = (tmp << 1) + block_tid;
			for (j = i + 1; j < edgeidx; j++)
			{
				py2 = RHash[j][block_tid];
				if (py2 == py)
				{
					RHash[j][block_tid] = 0xffffffff;
					gRHash[(SolveEN << 1) *block_id + (j << 1) + block_tid] = (tmp << 1) + block_tid;
				}
			}
			tmp++;
		}
	}
	__syncthreads();

}

__global__ void solve128X_127EN(uint32_t *gRege, uint8_t *gRHash, uint32_t *gproof)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t i, tmp;
	uint8_t u, v;

	uint32_t block_tid = id % SolveThreadsPerBlock;
	uint32_t *Rege = gRege + id * SolveEN;
	uint8_t *RHash = gRHash + id * (SolveEN << 1);
	uint32_t *proof = gproof + id * CLEN;

	__shared__ uint32_t path[SolveThreadsPerBlock][CLEN];
	__shared__ uint8_t graph[SolveThreadsPerBlock][SolveEN << 1];

	uint8_t pre;
	uint8_t cur;
	uint8_t next;

	memset(&graph[block_tid], 0xff, (SolveEN << 1));
	proof[0] = 0xffffffff;

	for (i = 0; i<SolveEN; i++)
	{
		if (Rege[i] == 0xffffffff)
		{
			break;
		}
		u = RHash[i << 1];
		v = RHash[(i << 1) + 1];
		__syncthreads();
		pre = 0xff;
		cur = u;
		while (cur != 0xff)
		{
			next = graph[block_tid][cur];
			graph[block_tid][cur] = pre;
			pre = cur;
			cur = next;
		}
		int m = 0;
		cur = v;
		while (graph[block_tid][cur] != 0xff && m < CLEN)
		{
			cur = graph[block_tid][cur];
			++m;
		}
		if (cur != u)
		{
			graph[block_tid][u] = v;
		}
		else if (m == CLEN - 1)
		{
			int j;
			cur = v;
			for (j = 0; j <= m; ++j)
			{
				path[block_tid][j] = cur;
				cur = graph[block_tid][cur];
			}

			memset(&graph[block_tid], 0xff, (SolveEN << 1));

			for (j = 1; j <= m; ++j)
			{
				graph[block_tid][path[block_tid][j]] = path[block_tid][j - 1];
			}

			int k = 0;
			int b = CLEN - 1;
			for (j = 0; k < b; ++j)
			{
				u = RHash[j << 1];
				v = RHash[(j << 1) + 1];
				if (graph[block_tid][u] == v || graph[block_tid][v] == u)
				{
					path[block_tid][k] = Rege[j];
					++k;
				}
			}
			path[block_tid][k] = Rege[i];

			for (j = 0; j < CLEN - 1; j++) // sort
			{
				for (k = 0; k < CLEN - j - 1; k++)
				{
					if (path[block_tid][k]>path[block_tid][k + 1])
					{
						tmp = path[block_tid][k];
						path[block_tid][k] = path[block_tid][k + 1];
						path[block_tid][k + 1] = tmp;
					}
				}
			}
			for (j = 0; j < CLEN; j++)proof[j] = path[block_tid][j];
			break;
		}
	}
	__syncthreads();
}

int gpu_cuckoo(uint32_t gpuid)
{
	if (CuckooNum > MaxCuckooNum) {
		printf("CuckooNum out of bound!!!\n");
		return 0;
	}

	if (CuckooNum % SolveThreadsPerBlock != 0) {
		printf("CuckooNum must be a multiple of SolveThreadsPerBlock = %5d\n", SolveThreadsPerBlock);
		return 0;
	}

	//alloc once
	if (gpu_divices[gpuid]->gmsg == NULL) {
		if (hipMalloc((void **)&gpu_divices[gpuid]->gmsg, CuckooNum * 32 * sizeof(uint8_t)) != hipSuccess) {
			printf("gmsg hipMalloc error\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->gRege == NULL) {
		if (hipMalloc((void **)&gpu_divices[gpuid]->gRege, CuckooNum * 128 * sizeof(uint32_t)) != hipSuccess) {
			printf("gRege hipMalloc error\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->gRHash == NULL) {
		if (hipMalloc((void **)&gpu_divices[gpuid]->gRHash, CuckooNum * 2 * 128 * sizeof(uint8_t)) != hipSuccess) {
			printf("gRHash hipMalloc error\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->gproof == NULL) {
		if (hipMalloc((void **)&gpu_divices[gpuid]->gproof, CuckooNum * CLEN * sizeof(uint32_t)) != hipSuccess) {
			printf("gproof hipMalloc error\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->gnode == NULL) {
		if (hipMalloc((void **)&gpu_divices[gpuid]->gnode, CuckooNum * M * sizeof(uint32_t)) != hipSuccess) {
			printf("gnode hipMalloc error\n");
			return 0;
		}
	}

	if (hipMemcpy(gpu_divices[gpuid]->gmsg, gpu_divices[gpuid]->msg, CuckooNum * 32 * sizeof(uint8_t), hipMemcpyHostToDevice) != hipSuccess) {
		printf("copy memory error\n");
		return 0;
	}

	kill_leaf << <CuckooNum, threadsPerBlock >> >(gpu_divices[gpuid]->gmsg, gpu_divices[gpuid]->gRege, gpu_divices[gpuid]->gRHash, gpu_divices[gpuid]->gnode);
	hipDeviceSynchronize();

	solve128X_127EN << <CuckooNum / SolveThreadsPerBlock, SolveThreadsPerBlock >> >(gpu_divices[gpuid]->gRege, gpu_divices[gpuid]->gRHash, gpu_divices[gpuid]->gproof);
	hipDeviceSynchronize();

	if (hipMemcpy(gpu_divices[gpuid]->cproof, gpu_divices[gpuid]->gproof, CuckooNum * CLEN * sizeof(uint32_t), hipMemcpyDeviceToHost) != hipSuccess) {
		printf("copy memory error\n");
		return 0;
	}
	return CuckooNum;
}

GPU_DEVICE* New_GPU_DEVICE()
{
	GPU_DEVICE* p = NULL;
	p = (GPU_DEVICE*)malloc(sizeof(GPU_DEVICE));
	if (p != NULL)
	{
		p->gmsg = NULL;
		p->gRHash = NULL;
		p->gRege = NULL;
		p->gproof = NULL;
		p->gnode = NULL;
	}
	return p;
}
void GPU_Count()
{
	int num;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&num);
	printf("deviceCount := %d\n", num);
	gpu_divices_cnt = 0;
	for (int i = 0; i<num; i++)
	{

		hipGetDeviceProperties(&prop, i);
		printf("name: %s\n", prop.name);
		printf("totalGlobalMem: %luG\n", prop.totalGlobalMem / 1024 / 1024 / 1024);
		printf("multiProcessorCount: %d\n", prop.multiProcessorCount);
		printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
		printf("major:%d,minor: %d\n", prop.major, prop.minor);
		gpu_divices_cnt++;
	}
}

extern "C" {
	int c_solve(uint32_t *prof, uint64_t *nonc, const uint8_t *hash, const uint8_t *target, uint32_t gpuid) {
		while (!gpu_divices[gpuid]) {
			gpu_divices[gpuid] = New_GPU_DEVICE();
		}

		uint8_t pmesg[CN];
		uint8_t thash[32];
		blake2b_state S;
		
		b2setup(&S);
		memcpy(pmesg+8, hash, 32);

		for(int i=0; i< CuckooNum; ++i) {
			RAND_bytes(pmesg, 8);
			blake2b_state tmp = S;
			blake2b_update(&tmp, pmesg, 40);
			blake2b_final(&tmp, gpu_divices[gpuid]->msg[i], 32);
			gpu_divices[gpuid]->nonces[i] = le64toh(((uint64_t *)pmesg)[0]);
		}
		
		hipSetDevice(gpuid);
		int ret = gpu_cuckoo(gpuid);

		for(int i=0; i< ret; ++i) {
			if (gpu_divices[gpuid]->cproof[i][0] != 0xffffffff)
			{
				memcpy(pmesg, gpu_divices[gpuid]->cproof[i], CN);
				blake2b_state tmp = S;
				blake2b_update(&tmp, pmesg, CN);
				blake2b_final(&tmp, thash, 32);

				for(int j=0; j<32; ++j) {
					if(thash[j] < target[j]) {
						memcpy(prof, gpu_divices[gpuid]->cproof[i], CN);
						*nonc = gpu_divices[gpuid]->nonces[i];
						prof[CLEN] = 1;
						return ret;
					} else if(thash[j] > target[j]) {
						break;
					}
				}
			}
		}

		return ret;
	}
}
